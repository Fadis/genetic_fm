#include "hip/hip_runtime.h"
#include <cstdint>
#include <vector>
#include <memory>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "fft.hpp"

void init_fft() {
}

#define checkCudaErrors( expr, exception ) \
{ \
  auto cuda_result = expr; \
  if( cuda_result != hipSuccess ) { \
    std::cout << __FILE__ << " " << __LINE__ << std::endl; \
    throw exception ( hipGetErrorString( hipGetLastError() ) );  \
  }\
}

#define checkCuFFTErrors( expr, exception ) \
{ \
  auto cuda_result = expr; \
  if( cuda_result != HIPFFT_SUCCESS ) { \
    std::cout << __FILE__ << " " << __LINE__ << " " << int( cuda_result ) << std::endl; \
    throw exception ( hipGetErrorString( hipGetLastError() ) ); \
  } \
}

struct fft_detail {
  float *window;
  size_t resolution;
  size_t interval;
  size_t width;
  const uint8_t *reference;
  size_t reference_batch_count;
  float diff;
  float *envelope;
  size_t batch_offset;
};

__device__ hipfftReal input_cb(
  void *src, 
  size_t offset, 
  void *callerInfo, 
  void *sharedPtr
) {
  fft_detail *detail = (fft_detail*)callerInfo;
  size_t index = offset % detail->resolution;
  size_t batch = detail->batch_offset + offset / detail->resolution;
  int16_t element = ((int16_t*)src)[ index + batch * detail->interval ];
  return ( hipfftReal )( element/32767.f * detail->window[ index ] );
}

__device__ void output_cb(
  void *dataOut, 
  size_t offset, 
  hipfftComplex element, 
  void *callerInfo, 
  void *sharedPtr
) {
  fft_detail *detail = (fft_detail*)callerInfo;
  size_t index = offset % ( (detail->resolution/2) + 1 );
  size_t batch = detail->batch_offset + offset / ( (detail->resolution/2) + 1 );
  if( index < detail->width ) {
    float abs = hipCabsf( element );
    atomicAdd( detail->envelope + batch, abs );
    ( (float*)dataOut )[ index + detail->width * batch ] = abs;
  }
}

__device__ void output_log_cb(
  void *dataOut, 
  size_t offset, 
  hipfftComplex element, 
  void *callerInfo, 
  void *sharedPtr
) {
  fft_detail *detail = (fft_detail*)callerInfo;
  size_t index = offset % ( (detail->resolution/2) + 1 );
  size_t batch = detail->batch_offset + offset / ( (detail->resolution/2) + 1 );
  if( index < detail->width ) {
    const float value = hipCabsf( element );
    atomicAdd( detail->envelope + batch, value );
    const float log_value_ = 80.f * __log10f( value < 1.f ? 1.f : value );
    const int log_value = log_value_ > 255.f ? int( 255 ) : int( log_value_ );
    ( (uint8_t*)dataOut )[ index + detail->width * batch ] = log_value;
  }
}

__device__ void output_comp_cb(
  void *dataOut, 
  size_t offset, 
  hipfftComplex element, 
  void *callerInfo, 
  void *sharedPtr
) {
  fft_detail *detail = (fft_detail*)callerInfo;
  const size_t index = offset % ( (detail->resolution/2) + 1 );
  const size_t batch = detail->batch_offset + offset / ( (detail->resolution/2) + 1 );
  if( index < detail->width ) {
    if( batch < detail->reference_batch_count ) {
      const float value = hipCabsf( element );
      atomicAdd( detail->envelope + batch, value );
      const float log_value_ = 80.f * __log10f( value < 1.f ? 1.f : value );
      const int log_value = log_value_ > 255.f ? int( 255 ) : int( log_value_ );
      atomicAdd( &detail->diff, float( __sad( log_value, int( detail->reference[ index + detail->width * batch ] ), 0 ) ) );
    }
    else {
      const float value = hipCabsf( element );
      atomicAdd( detail->envelope + batch, value );
      const float log_value_ = 80.f * __log10f( value < 1.f ? 1.f : value );
      const float log_value = log_value_ > 255.f ? 255.f : float( log_value_ );
      atomicAdd( &detail->diff, log_value );
    }
  }
}


__device__ hipfftCallbackLoadR input_cb_ptr_d = input_cb; 
__device__ hipfftCallbackStoreC output_cb_ptr_d = output_cb;
__device__ hipfftCallbackStoreC output_log_cb_ptr_d = output_log_cb;
__device__ hipfftCallbackStoreC output_comp_cb_ptr_d = output_comp_cb;

__global__ void generate_window( fft_detail *detail ) {
  size_t index = threadIdx.x + blockIdx.x * 1024;
  detail->window[ index ] = __sinf( float( M_PI ) * float( index ) / detail->resolution ); 
}

__global__ void generate_window( float *window, unsigned int resolution ) {
  size_t index = threadIdx.x + blockIdx.x * 1024;
  window[ index ] = __sinf( float( M_PI ) * float( index ) / resolution ); 
}

__global__ void add_lacking_batches( fft_detail *detail, size_t offset ) {
  size_t index = threadIdx.x + blockIdx.x * 1024u + offset;
  atomicAdd( &detail->diff, float( detail->reference[ index ] ) );
}

using window_list_t = boost::container::flat_map< unsigned int, std::shared_ptr< float > >;

window_list_t generate_window() {
  window_list_t result;
  for( unsigned int i = 16u; i != 65536u; i <<= 1 ) {
    float *window;
    checkCudaErrors( hipMalloc( &window, sizeof(float)*i ), fft_allocation_failed );
    std::shared_ptr< float > wrapped( window, &hipFree );
    if( i > 1024 ) generate_window<<< i/1024, 1024 >>>( window, i );
    else generate_window<<< 1, i >>>( window, i );
    result.insert( result.end(), std::make_pair( i, wrapped ) );
  }
  checkCudaErrors( hipDeviceSynchronize(), fft_initialization_failed );
  return std::move( result );
}

std::shared_ptr< float > fft( const window_list_t &window, const std::vector< int16_t > &data, size_t resolution, size_t interval, size_t width ) {
  const auto window_iter = window.find( resolution );
  if( window_iter == window.end() ) throw fft_initialization_failed( "invalid resolution" );
  const size_t batch = ( data.size() - resolution )/interval + 1;
  float *envelope;
  checkCudaErrors(hipMalloc( &envelope, sizeof(float)*batch), fft_allocation_failed );
  std::shared_ptr< float > wrapped_envelope( envelope, &hipFree );
  checkCudaErrors( hipMemset( envelope, 0, batch * sizeof(float) ), fft_initialization_failed );
  fft_detail *detail;
  checkCudaErrors(hipMallocManaged( &detail, sizeof(fft_detail),hipMemAttachGlobal), fft_allocation_failed );
  std::shared_ptr< fft_detail > wrapped_detail( detail, &hipFree );
  detail->resolution = resolution;
  detail->interval = interval;
  detail->width = width;
  detail->window = window_iter->second.get();
  detail->envelope = envelope;
  detail->batch_offset = 0u;
  int16_t *input;
  checkCudaErrors(hipMalloc( &input, sizeof(int16_t)*interval*batch), fft_allocation_failed );
  std::shared_ptr< int16_t > wrapped_input( input, &hipFree );
  checkCudaErrors(hipMemcpy( input, data.data(), sizeof(int16_t)*data.size(), hipMemcpyHostToDevice ), fft_data_transfar_failed );
  checkCudaErrors(hipMemset( input + sizeof(int16_t)*data.size(), 0, sizeof( int16_t )*( interval*batch - data.size() ) ), fft_data_transfar_failed );
  float *output;
  checkCudaErrors(hipMalloc( &output, sizeof(float)*width*batch), fft_allocation_failed );
  std::shared_ptr< float > wrapped_output( output, &hipFree );
  hipfftHandle plan;
  checkCuFFTErrors( hipfftCreate( &plan ), fft_initialization_failed );
  int signal_size = resolution;
  size_t work_size;
  checkCuFFTErrors( hipfftMakePlanMany( plan, 1, &signal_size, 0, 0, 0, 0, 0, 0, HIPFFT_R2C, batch, &work_size ), fft_initialization_failed );
  hipfftCallbackLoadR input_cb_ptr_h;
  checkCudaErrors( hipMemcpyFromSymbol( &input_cb_ptr_h, HIP_SYMBOL(input_cb_ptr_d), sizeof( hipfftCallbackLoadR ) ), fft_data_transfar_failed );
  hipfftCallbackStoreC output_cb_ptr_h;
  checkCudaErrors( hipMemcpyFromSymbol( &output_cb_ptr_h, HIP_SYMBOL(output_cb_ptr_d), sizeof( hipfftCallbackStoreC ) ), fft_data_transfar_failed );
  checkCuFFTErrors( hipfftXtSetCallback( plan, (void **)&input_cb_ptr_h, HIPFFT_CB_LD_REAL, (void **)&detail ), fft_initialization_failed );
  checkCuFFTErrors( hipfftXtSetCallback( plan, (void **)&output_cb_ptr_h, HIPFFT_CB_ST_COMPLEX, (void **)&detail ), fft_initialization_failed );
  checkCuFFTErrors( hipfftExecR2C( plan, (hipfftReal*)input, (hipfftComplex *)output ), fft_execution_failed );
  checkCudaErrors( hipDeviceSynchronize(), fft_execution_failed );
  return std::move( wrapped_output );
}

std::pair< std::vector< float >, std::shared_ptr< uint8_t > > fftref( const window_list_t &window, const std::vector< int16_t > &data, size_t resolution, size_t interval, size_t width ) {
  const auto window_iter = window.find( resolution );
  if( window_iter == window.end() ) throw fft_initialization_failed( "invalid resolution" );
  const size_t batch = data.size() > resolution ? ( data.size() - resolution )/interval + 1 : 1u;
  float *envelope_d;
  checkCudaErrors(hipMalloc( &envelope_d, sizeof(float)*batch), fft_allocation_failed );
  std::shared_ptr< float > wrapped_envelope( envelope_d, &hipFree );
  checkCudaErrors( hipMemset( envelope_d, 0, batch * sizeof(float) ), fft_initialization_failed );
  fft_detail *detail;
  checkCudaErrors(hipMallocManaged( &detail, sizeof(fft_detail),hipMemAttachGlobal), fft_allocation_failed );
  std::shared_ptr< fft_detail > wrapped_detail( detail, &hipFree );
  detail->resolution = resolution;
  detail->interval = interval;
  detail->width = width;
  detail->window = window_iter->second.get();
  detail->envelope = envelope_d;
  detail->batch_offset = 0u;
  int16_t *input;
  size_t input_size = std::max( interval*batch, data.size() );
  checkCudaErrors(hipMalloc( &input, sizeof(int16_t)*input_size), fft_allocation_failed );
  std::shared_ptr< int16_t > wrapped_input( input, &hipFree );
  checkCudaErrors(hipMemcpy( input, data.data(), sizeof(int16_t)*data.size(), hipMemcpyHostToDevice ), fft_data_transfar_failed );
  if( input_size > data.size() ) {
    checkCudaErrors( hipMemset( input + data.size(), 0, sizeof( int16_t )*( input_size - data.size() ) ), fft_initialization_failed );
  }
  uint8_t *output;
  checkCudaErrors(hipMalloc( &output, sizeof(uint8_t)*width*batch), fft_allocation_failed );
  std::shared_ptr< uint8_t > wrapped_output( output, &hipFree );
  hipfftCallbackLoadR input_cb_ptr_h;
  checkCudaErrors( hipMemcpyFromSymbol( &input_cb_ptr_h, HIP_SYMBOL(input_cb_ptr_d), sizeof( hipfftCallbackLoadR ) ), fft_data_transfar_failed );
  hipfftCallbackStoreC output_cb_ptr_h;
  checkCudaErrors( hipMemcpyFromSymbol( &output_cb_ptr_h, HIP_SYMBOL(output_log_cb_ptr_d), sizeof( hipfftCallbackStoreC ) ), fft_data_transfar_failed );
  for( size_t batch_offset = 0u; batch_offset < batch; batch_offset += 4200u ) {
    std::cout << batch << " " << batch_offset << " " << std::min( batch - batch_offset, size_t( 4200u ) ) << std::endl;
    detail->batch_offset = batch_offset;
    hipfftHandle plan;
    checkCuFFTErrors( hipfftCreate( &plan ), fft_initialization_failed );
    int signal_size = resolution;
    size_t work_size;
    checkCuFFTErrors( hipfftMakePlanMany( plan, 1, &signal_size, 0, 0, 0, 0, 0, 0, HIPFFT_R2C, std::min( batch - batch_offset, size_t( 4200u ) ), &work_size ), fft_initialization_failed );
    checkCuFFTErrors( hipfftXtSetCallback( plan, (void **)&input_cb_ptr_h, HIPFFT_CB_LD_REAL, (void **)&detail ), fft_initialization_failed );
    checkCuFFTErrors( hipfftXtSetCallback( plan, (void **)&output_cb_ptr_h, HIPFFT_CB_ST_COMPLEX, (void **)&detail ), fft_initialization_failed );
    checkCuFFTErrors( hipfftExecR2C( plan, (hipfftReal*)input, (hipfftComplex *)output ), fft_execution_failed );
    checkCudaErrors( hipDeviceSynchronize(), fft_execution_failed );
    checkCuFFTErrors( hipfftDestroy( plan ), fft_allocation_failed );
  }
  std::vector< float > envelope_h( batch );
  checkCudaErrors( hipMemcpy( envelope_h.data(), detail->envelope, sizeof(float)*batch, hipMemcpyDeviceToHost ), fft_data_transfar_failed );
  return std::make_pair( std::move( envelope_h ), std::move( wrapped_output ) );
}

std::pair< float, std::vector< float > > fftcomp( const uint8_t *ref, size_t reference_batch_count, const window_list_t &window, const std::vector< int16_t > &data, size_t resolution, size_t interval, size_t width ) {
  const auto window_iter = window.find( resolution );
  if( window_iter == window.end() ) throw fft_initialization_failed( "invalid resolution" );
  const size_t batch = data.size() > resolution ? ( data.size() - resolution )/interval + 1 : 1u;
  float *envelope_d;
  checkCudaErrors(hipMalloc( &envelope_d, sizeof(float)*batch), fft_allocation_failed );
  std::shared_ptr< float > wrapped_envelope( envelope_d, &hipFree );
  checkCudaErrors( hipMemset( envelope_d, 0, batch * sizeof(float) ), fft_initialization_failed );
  fft_detail *detail;
  checkCudaErrors(hipMallocManaged( &detail, sizeof(fft_detail),hipMemAttachGlobal), fft_allocation_failed );
  std::shared_ptr< fft_detail > wrapped_detail( detail, &hipFree );
  detail->resolution = resolution;
  detail->interval = interval;
  detail->width = width;
  detail->window = window_iter->second.get();
  detail->envelope = envelope_d;
  detail->reference = ref;
  detail->reference_batch_count = reference_batch_count;
  detail->diff = 0.0f;
  detail->batch_offset = 0u;
  int16_t *input;
  size_t input_size = std::max( data.size(), batch*interval );
  checkCudaErrors(hipMallocManaged( &input, sizeof(int16_t)*input_size ), fft_allocation_failed );
  std::shared_ptr< int16_t > wrapped_input( input, &hipFree );
  checkCudaErrors(hipMemcpy( input, data.data(), sizeof(int16_t)*data.size(), hipMemcpyHostToDevice ), fft_data_transfar_failed );
  if( input_size > data.size() ) {
    checkCudaErrors( hipMemset( input + data.size(), 0, sizeof( int16_t )*( input_size - data.size() ) ), fft_initialization_failed );
  }
  hipfftCallbackLoadR input_cb_ptr_h;
  checkCudaErrors( hipMemcpyFromSymbol( &input_cb_ptr_h, HIP_SYMBOL(input_cb_ptr_d), sizeof( hipfftCallbackLoadR ) ), fft_data_transfar_failed );
  hipfftCallbackStoreC output_cb_ptr_h;
  checkCudaErrors( hipMemcpyFromSymbol( &output_cb_ptr_h, HIP_SYMBOL(output_comp_cb_ptr_d), sizeof( hipfftCallbackStoreC ) ), fft_data_transfar_failed );
  for( size_t batch_offset = 0u; batch_offset < batch; batch_offset += 4200u ) {
    detail->batch_offset = batch_offset;
    hipfftHandle plan;
    checkCuFFTErrors( hipfftCreate( &plan ), fft_initialization_failed );
    int signal_size = resolution;
    size_t work_size;
    checkCuFFTErrors( hipfftMakePlanMany( plan, 1, &signal_size, 0, 0, 0, 0, 0, 0, HIPFFT_R2C, std::min( batch - batch_offset, size_t( 4200u ) ), &work_size ), fft_initialization_failed );
    checkCuFFTErrors( hipfftXtSetCallback( plan, (void **)&input_cb_ptr_h, HIPFFT_CB_LD_REAL, (void **)&detail ), fft_initialization_failed );
    checkCuFFTErrors( hipfftXtSetCallback( plan, (void **)&output_cb_ptr_h, HIPFFT_CB_ST_COMPLEX, (void **)&detail ), fft_initialization_failed );
    checkCuFFTErrors( hipfftExecR2C( plan, (hipfftReal*)input, (hipfftComplex *)nullptr ), fft_execution_failed );
    checkCudaErrors( hipDeviceSynchronize(), fft_execution_failed );
    checkCuFFTErrors( hipfftDestroy( plan ), fft_allocation_failed );
  }
  if( batch < reference_batch_count ) {
    size_t left_count = ( reference_batch_count - batch ) * width;
    size_t left_block = left_count / 1024u;
    size_t left_mod = left_count % 1024u;
    if( left_block )
      add_lacking_batches<<< left_block, 1024u >>>( detail, size_t( batch * width ) );
    if( left_mod )
      add_lacking_batches<<< 1u, left_mod >>>( detail, size_t( batch * width + left_block * 1024u ) );
  }
  checkCudaErrors( hipDeviceSynchronize(), fft_execution_failed );
  std::vector< float > envelope_h( batch );
  checkCudaErrors( hipMemcpy( envelope_h.data(), detail->envelope, sizeof(float)*batch, hipMemcpyDeviceToHost ), fft_data_transfar_failed );
  return std::make_pair( detail->diff, std::move( envelope_h ) );

}

